#include "hip/hip_runtime.h"
// Originally written by Supranational LLC

#include <iostream>
#include <fstream>
#include <cstdint>
#define FEATURE_BLS12_381
#include <ff/bls12-381.hpp>
#include <util/gpu_t.cuh>
#include <util/rusterror.h>
#include <ntt/ntt.cuh>
#include <errno.h>

const uint32_t PARAM_EXT = 6;
const uint32_t PARAM_M = 1u << PARAM_EXT;
const uint32_t PARAM_B = 119;
const uint32_t PARAM_C = 127;
const uint32_t PARAM_BC = 15113; // PARAM_B * PARAM_C
const size_t PAGE_SIZE = 4096;

#include "common_operations.cuh"
#include "chacha.cuh"
#include "data_structures.cuh"
#include "sort15113.cuh"
#include "blake3.cuh"
#include "create_tables.cuh"
#include "find_proof.cuh"
#include "find_matches.cuh"

// Returns the number of available GPUs
extern "C" {
    size_t gpu_count() {
        return ngpus();
    }
}

template<int K>
void create_tables(gpu_specific_state& gpu_state, const chacha_state& state,
                   uint32_t output_block_count, uint32_t table_size, size_t shared_sz,
                   size_t shared_sz_matches, const int y_bit_size, uint32_t nbuckets,
                   uint32_t* d_match_counter)
{
    const gpu_t& gpu = gpu_state.gpu;

    //------------------------First Table-----------------------
    // Generates the ChaCha key stream for the first table.
    generate_chacha_keystream<8><<<gpu.sm_count(), 1024, 0, gpu>>>
        (gpu_state.out_chacha, state, output_block_count);

    CUDA_OK(hipGetLastError());

    // Computes the `y` values for the first table.
    compute_f1<K, PARAM_EXT><<<gpu.sm_count(), 1024, 0, gpu>>>
        (gpu_state.tables[0].y, (uint32_t*)&gpu_state.out_chacha[0], table_size);

    CUDA_OK(hipGetLastError());

    // Sorts the first table.
    gpu.launch_coop(sort15113, launch_params_t{gpu.sm_count(), SORT_BLOCKDIM, shared_sz},
                    &gpu_state.tables[0].ys[0],
                    (const uint32_t*)gpu_state.tables[0].y,
                    table_size, &gpu_state.temp_sort[0],
                    &gpu_state.histogram[0], false, y_bit_size);

    //------------------------Other Tables-----------------------
    for (size_t i = 1; i < NUM_TABLES; i++) {
        // Resets the match counter for the current table.
        gpu.bzero(&d_match_counter[0], 1);
        metadata* metadatas_input = gpu_state.tables[i - 1].metadatas;

        // Finds matches for the current table.
        find_matches<<<nbuckets - 1, block_sz, shared_sz_matches * sizeof(uint32_t), gpu>>>
            (&gpu_state.tables[i].matches[0], gpu_state.tables[i - 1].ys,
             &gpu_state.histogram[0], &d_match_counter[0]);

        CUDA_OK(hipGetLastError());

        // Computes the `y` for the current table.
        compute_fn<K, PARAM_EXT><<<gpu.sm_count(), 1024, 0, gpu>>>
            (gpu_state.tables[i].y, gpu_state.tables[i].metadatas,
             gpu_state.tables[i - 1].ys, gpu_state.tables[i - 1].y,
             gpu_state.tables[i].matches, &d_match_counter[0],
             metadatas_input, i + 1);

        CUDA_OK(hipGetLastError());

        // Sorts the current table.
        gpu.launch_coop(sort15113, launch_params_t{gpu.sm_count(), SORT_BLOCKDIM, shared_sz},
                        &gpu_state.tables[i].ys[0], (const uint32_t*)gpu_state.tables[i].y,
                        (const uint32_t*)&d_match_counter[0],
                        &gpu_state.temp_sort[0], &gpu_state.histogram[0],
                        i == NUM_TABLES - 1, y_bit_size);
    }
}

// This function generates and encodes PoSpace on the GPU,
// specifically calculating the chunks_scratch vector.
template <uint32_t K>
RustError::by_value generate_and_encode_pospace(const uint8_t* key,
                                                uint32_t lg_record_size,
                                                uint32_t* challenge_index,
                                                const fr_t* record,
                                                uint8_t* chunks_scratch,
                                                uint32_t* proof_count,
                                                fr_t* parity_record_chunks,
                                                int gpu_id)
{
    const int y_bit_size = K + PARAM_EXT;

    uint32_t record_size = 1 << lg_record_size;
    uint32_t challenge_len = record_size << 1;

    size_t output_len_bytes = K * ((size_t)1 << K) / 8;
    size_t output_block_count = output_len_bytes / 64;

    uint32_t table_size = 1 << K;
    size_t shared_sz = sizeof(uint32_t) << DIGIT_BITS;

    // nonce is initialized as a 12-byte array with all zeros.
    uint8_t nonce[] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

    // Initializes the ChaCha state with the key and nonce.
    chacha_state state = initialize_chacha_state<uint32_t>(key, nonce);

    uint32_t nbuckets = (1u << y_bit_size) / PARAM_BC + 1;
    size_t shared_sz_matches = (NUM_TABLES - 1) * block_sz * sizeof(uint2) / sizeof(uint32_t) + 512;

    // Gets the GPU-specific state for the given GPU_ID.
    gpu_specific_state& gpu_state = proof_state.gpu_specific_states[gpu_id];
    // Locks the mutex for the specific GPU identified by GPU_ID.
    std::lock_guard<std::mutex> lock(*gpu_state.mtx);

    // Gets the GPU object.
    const gpu_t& gpu = select_gpu(gpu_id);

    semaphore_t complete;
    gpu.spawn([=, &complete]() {
        struct vm {
            static void touch(const void *ptr, size_t len)
            {
                auto* data = reinterpret_cast<const volatile char*>(ptr);
                for (size_t i = 0; i < len; i += PAGE_SIZE) {
                    data[i];
                }
                data[len - 1];
            }
        };

        vm::touch(parity_record_chunks, record_size * sizeof(fr_t));
        vm::touch(challenge_index, challenge_len * sizeof(uint32_t));
        vm::touch(chunks_scratch, challenge_len * 32);
        complete.notify();
    });

    try {
        // An object to help us sync different asynchronous streams
        event_t sync_event;

        // Allocates and initializes various data on the GPU.

        // Initializes the GPU state.
        gpu_state.initialize_data(K, challenge_len);

        dev_ptr_t<uint32_t> d_match_counter(1, gpu);
        gpu.bzero(&d_match_counter[0], 1);

        dev_ptr_t<uint32_t> d_proof_counter(1, gpu);
        gpu.bzero(&d_proof_counter[0], 1);

        table_data_ptr tables_data[NUM_TABLES];
        for (uint32_t i = 0; i < NUM_TABLES; i++) {
            tables_data[i] = gpu_state.tables[i].get_table_data();
        }

        dev_ptr_t<table_data_ptr> d_tables(NUM_TABLES, gpu);
        gpu.HtoD(&d_tables[0], tables_data, NUM_TABLES);

        dev_ptr_t<uint32_t> d_chunks_scratch(challenge_len * 8);

        dev_ptr_t<fr_t> d_record{2 * record_size, gpu};
        gpu.HtoD(&d_record[record_size], record, record_size);

        // End of GPU data allocation and initialization.

        // Convert the inputs from big-endian into little-endian
        kern_endianness_swap<fr_t><<<gpu.sm_count(), 1024, 0, gpu>>>(&d_record[record_size],
                                                                     &d_record[record_size],
                                                                     record_size);

        CUDA_OK(hipGetLastError());

        // Converts and duplicates data on the GPU.
        convert_to_mont<fr_t><<<gpu.sm_count(), 1024, 0, gpu>>>(&d_record[record_size],
                                                                &d_record[record_size],
                                                                record_size);

        CUDA_OK(hipGetLastError());

        // Performs inverse NTT on the data in d_record with a given lg_record_size.
        NTT::Base_dev_ptr(gpu, &d_record[record_size], lg_record_size,
                          NTT::InputOutputOrder::NR, NTT::Direction::inverse,
                          NTT::Type::standard);

        NTT::LDE_expand(gpu, &d_record[0], &d_record[record_size], lg_record_size, 1);

        // Performs forward NTT on the data in d_record with a (lg_record_size + 1).
        NTT::Base_dev_ptr(gpu, d_record, lg_record_size + 1,
                          NTT::InputOutputOrder::RN, NTT::Direction::forward,
                          NTT::Type::standard);

        // Convert the inputs (and outputs) back to big-endian
        kern_endianness_swap<fr_t><<<gpu.sm_count(), 1024, 0, gpu>>>(&d_record[0],
                                                                     &d_record[0],
                                                                     2 * record_size);

        CUDA_OK(hipGetLastError());

        // Record a list of pending operations submitted to the stream
        sync_event.record(gpu);

        // Creates all NUM_TABLES tables
        create_tables<K>(gpu_state, state, output_block_count, table_size,
                         shared_sz, shared_sz_matches, y_bit_size, nbuckets,
                         d_match_counter);

        // Finds proof and creates chunks scratch
        create_chunks_scratch<K, PARAM_EXT><<<gpu.sm_count(), 256, 0, gpu>>>
            (d_chunks_scratch, &d_proof_counter[0], gpu_state.challenge_index,
             challenge_len, d_record, d_tables, &d_match_counter[0]);

        CUDA_OK(hipGetLastError());

        // Wait for the list of operations which were recorded to complete
        // before launching any operations on stream gpu[0]
        // The goal is to overlap the next memory copy with create table and find proof kernels
        sync_event.wait(gpu[0]);

        // Creates parity record chunks by transferring odd-indexed elements.
        gpu[0].DtoH(parity_record_chunks, &d_record[1], record_size, 2*sizeof(fr_t));

        // Transfers the proof count from the GPU to the host.
        gpu.DtoH(&proof_count[0], &d_proof_counter[0], 1);

        // Transfers the challenge indices from the GPU to the host.
        gpu.DtoH(challenge_index, gpu_state.challenge_index, proof_count[0]);

        // Transfers the chunks scratch data from the GPU to the host.
        gpu.DtoH((uint32_t*)chunks_scratch, d_chunks_scratch, proof_count[0] * 8);

        // Automatically syncs gpu[0] as well
        gpu.sync();
        complete.wait();
    } catch (const cuda_error& e) {
        gpu.sync();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
#else
        return RustError{e.code()};
#endif
    }

    return RustError{hipSuccess};
}

// This function calls a templated version of `generate_and_encode_pospace` based on the value of K.
// The support for K values is limited to a maximum of 21 due to constraints in the sorting implementation.
// Specifically, the algorithm calculates an "upper digit" based on the bit length of the data being sorted, divided by PARAM_BC.
// The upper digit has a maximum allowable value of 2^14.
// As K increases, the bit length of the data also increases.
// When K reaches 22, the calculated upper digit exceeds the 2^14 limit.
// Consequently, to ensure correct operation, the algorithm is restricted to K values of 21 or less.
extern "C"
RustError::by_value generate_and_encode_pospace_dispatch(uint32_t K,
                                                         const uint8_t* key,
                                                         uint32_t lg_record_size,
                                                         uint32_t* challenge_index,
                                                         const fr_t* record,
                                                         uint8_t* chunks_scratch,
                                                         uint32_t* proof_count,
                                                         fr_t* parity_record_chunks,
                                                         int gpu_id)
{
    switch (K) {
        case 15: return generate_and_encode_pospace<15>(key, lg_record_size,
                                                        challenge_index, record,
                                                        chunks_scratch, proof_count,
                                                        parity_record_chunks, gpu_id);
        case 16: return generate_and_encode_pospace<16>(key, lg_record_size,
                                                        challenge_index, record,
                                                        chunks_scratch, proof_count,
                                                        parity_record_chunks, gpu_id);
        case 17: return generate_and_encode_pospace<17>(key, lg_record_size,
                                                        challenge_index, record,
                                                        chunks_scratch, proof_count,
                                                        parity_record_chunks, gpu_id);
        case 18: return generate_and_encode_pospace<18>(key, lg_record_size,
                                                        challenge_index, record,
                                                        chunks_scratch, proof_count,
                                                        parity_record_chunks, gpu_id);
        case 19: return generate_and_encode_pospace<19>(key, lg_record_size,
                                                        challenge_index, record,
                                                        chunks_scratch, proof_count,
                                                        parity_record_chunks, gpu_id);
        case 20: return generate_and_encode_pospace<20>(key, lg_record_size,
                                                        challenge_index, record,
                                                        chunks_scratch, proof_count,
                                                        parity_record_chunks, gpu_id);
        case 21: return generate_and_encode_pospace<21>(key, lg_record_size,
                                                        challenge_index, record,
                                                        chunks_scratch, proof_count,
                                                        parity_record_chunks, gpu_id);
        default: return RustError{EINVAL};
    }
}
